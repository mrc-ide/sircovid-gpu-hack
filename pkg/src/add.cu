#include "hip/hip_runtime.h"
#include "add.h"

__global__ void
vector_add(const double *A, const double *B, double *C, int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < numElements) {
    C[i] = A[i] + B[i];
  }
}

void add_gpu(double *A, double *B, double *C, int *n) {
  // Device Memory
  double *d_A, *d_B, *d_C;

  // Define the execution configuration
  dim3 blockSize(256, 1, 1);
  dim3 gridSize(1, 1, 1);
  gridSize.x = (*n + blockSize.x - 1) / blockSize.x;

  // Allocate output array
  hipMalloc((void**)&d_A, *n * sizeof(double));
  hipMalloc((void**)&d_B, *n * sizeof(double));
  hipMalloc((void**)&d_C, *n * sizeof(double));

  // copy data to device
  hipMemcpy(d_A, A, *n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, *n * sizeof(double), hipMemcpyHostToDevice);

  // GPU vector add
  vector_add<<<gridSize,blockSize>>>(d_A, d_B, d_C, *n);

  // Copy output
  hipMemcpy(C, d_C, *n * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
